#include "hip/hip_runtime.h"
#include "total_power.h"

extern "C" void get_total_power(unsigned char * input, float * output);

__global__ void total_power_kernel1(unsigned char * input, float * output) {

    // Declare dynamic shared memory
    __shared__ float power[pow1];

    // Get indicies
    int c = blockIdx.x;
    int f = blockIdx.y;
    int t = threadIdx.x;
    
    // Get internal index (internal to block)
    int sid = t;

    // Get absolute index
    int idx = f + 8*Nf*c + 8*Nf*Nc*t;

    if (sid < Nm*Nt) {
    	// Extract real and imaginary components;
    	float real = (float)input[2*idx];
    	float imag = (float)input[2*idx + 1];

    	// Compute instantaneous power
    	power[sid] = real*real + imag*imag;
    }
    else {
    	power[sid] = 0;
    }

    // Complete power computation before moving on
    __syncthreads();

    // Perform reduction
    for (int s = blockDim.x/2; s > 0; s>>=1) {
        if (sid < s) {
            power[sid] += power[sid + s];
        }
        __syncthreads();
    }

    // Save sum to output
    if (sid == 0) {
        output[f + 8*Nf*c] = power[0];
    }
}


__global__ void total_power_kernel2(float * input, float * output) {

    // Declare dynamic shared memory
    __shared__ float power[pow2];
    
    // Get indices
    int f = blockIdx.x;
    int c = threadIdx.x;

    // Get internal index (internal to block)
    int sid = c;

    // Get absolute index
    int idx = f + 8*Nf*c;

    if (sid < Nc) {
    	// Copy input to shared memory
    	power[sid] = input[idx];
    }
    else {
    	power[sid] = 0.0;
    }

    // Finish copy before proceeding
    __syncthreads();

    // Perform reduction
    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (sid < s) {
            power[sid] += power[sid + s];
        }
        __syncthreads();
    }

    // Save sum to input
    if (sid == 0) {
        output[f] = power[0];
    }
}


void get_total_power(unsigned char * input, float * output) {
    unsigned char * d_input;
    float * d_output1;
    float * d_output2;

    hipMalloc((void **) &d_input, 8*Nf*Nc*Nt*Nm*sizeof(unsigned char)*2);
    hipMalloc((void **) &d_output1, 8*Nf*Nc*sizeof(float));
    hipMalloc((void **) &d_output2, 8*Nf*sizeof(float));
    hipMemcpy(d_input, input, 8*Nf*Nc*Nt*Nm*2*sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 gridSize1(Nc,8*Nf,1);
    dim3 blockSize1(pow1,1);
    
    total_power_kernel1<<<gridSize1, blockSize1>>>(d_input, d_output1);
    hipError_t ret = hipGetLastError();
    if (ret != hipSuccess) {
        printf("ERROR: total_power_kernel1 - %s\n", hipGetErrorString(ret));
    }

    dim3 gridSize2(8*Nf,1,1);
    dim3 blockSize2(pow2,1,1);

    total_power_kernel2<<<gridSize2, blockSize2>>>(d_output1, d_output2);
    ret = hipGetLastError();
    if (ret != hipSuccess) {
        printf("ERROR: total_power_kernel2 - %s\n", hipGetErrorString(ret));
    }

    hipMemcpy(output, d_output2, 8*Nf*sizeof(float), hipMemcpyDeviceToHost);
}
