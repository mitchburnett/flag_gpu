#include "hip/hip_runtime.h"
#include "flag_beamformer.h"
#include <hip/hip_complex.h>

/*
__global__
void beamform(const hipDoubleComplex * data_in,
              const hipDoubleComplex * weights,
              float * data_out) {

	int f = blockIdx.x;
	int b = blockIdx.y;
	int t = threadIdx.x;
	int s = blockIdx.z;
	//int e = threadIdx.y;

	int i = input_idx(s*N_TIME_STI + t,f,0);//int i = input_idx(t,f,e);
	int w = weight_idx(b,f,0);//int w = weight_idx(b,f,e);
	//int c = sample_idx(t,b,f);
	// ************ Complex multiplication *************
	hipDoubleComplex elem_val;
	hipDoubleComplex weight_val;
	hipDoubleComplex new_beam;
	float beam_power;

	int e; // Element index
	float scale = 1.0/N_TIME_STI;
	//New variables//////
	__shared__ float reduced_array[N_STI_BLOC];
	//reduced_array = (float *)malloc(N_TBF*sizeof(float));
	/////////////////////
	new_beam.x = 0.0;
	new_beam.y = 0.0;

	if (t < N_TIME_STI) {

		for (e = 0; e < N_ELE; e++) { // Loop over elements
			elem_val = data_in[i+e];
			weight_val = weights[w+e];
			new_beam = hipCadd(new_beam, hipCmul(elem_val, hipConj(weight_val)));
		}

		beam_power = (float)(hipCmul(new_beam, hipConj(new_beam)).x);
		//beam_power = (float) (hipCmul(data_in[i], hipConj(weights[w])).x);
		atomicAdd(&data_out[output_idx(b,s,f)], beam_power*scale);
	}

	//New code///////////////////////////////////////////////

	//if(t<N_TIME_STI){
	//	reduced_array[t] = beam_power;
	//}
	//else{
	//	reduced_array[t] = 0.0;
	//}
	//__syncthreads();
	//
	//for(int k = blockDim.x/2; k>0; k>>=1){
	//	if(t<k){
	//		reduced_array[t] += reduced_array[t+k];
	//	}
	//	__syncthreads();
	//}
	//if(t == 0){
	//	data_out[output_idx(b,s,f)] = reduced_array[0]*scale;
	//}

	/////////////////////////////////////////////////////////
}
 */



__global__
void beamform(const unsigned char * data_in,
		const hipFloatComplex * weights,
		hipFloatComplex * beamformed) {

	int e = threadIdx.x;
	int t = blockIdx.x;
	int f = blockIdx.y;
	int b = blockIdx.z;
	//

	int i = input_idx(t,f,e);
	int w = weight_idx(b,f,e);
	//int c = sample_idx(t,b,f);
	// ************ Complex multiplication *************
	//hipDoubleComplex elem_val;
	//hipDoubleComplex weight_val;
	//hipDoubleComplex new_beam;

	//float scale = 1.0/N_ELE;
	//New variables//////
	__shared__ hipFloatComplex reduced_mul[N_ELE_BLOC];
	/////////////////////
	//new_beam.x = 0.0;
	//new_beam.y = 0.0;

	//elem_val = data_in[i];
	//weight_val = weights[w];
	//new_beam = hipCmul(elem_val, hipConj(weight_val));

	//New code///////////////////////////////////////////////

	if(e<N_ELE) {
		reduced_mul[e].x = data_in[2*i]   * weights[w].x + data_in[2*i+1] * weights[w].y;
		reduced_mul[e].y = data_in[2*i+1] * weights[w].x - data_in[2*i]   * weights[w].y;
	}
	else {
		reduced_mul[e].x = 0;
		reduced_mul[e].y = 0;
	}
	__syncthreads();

	//atomicAdd(&(beamformed[sample_idx(t,b,f)].x),reduced_mul[e].x);
	//atomicAdd(&(beamformed[sample_idx(t,b,f)].y),reduced_mul[e].y);

	for(int k = blockDim.x/2; k>0; k>>=1){
		if(e<k){
			//reduced_mul[e] = hipCaddf(reduced_mul[e], reduced_mul[e+k]);
			reduced_mul[e].x = reduced_mul[e].x + reduced_mul[e+k].x;
			reduced_mul[e].y = reduced_mul[e].y + reduced_mul[e+k].y;
		}
		__syncthreads();
	}
	if(e == 0){
		beamformed[sample_idx(t,b,f)] = reduced_mul[0];
	}


	/////////////////////////////////////////////////////////
}


__global__
void sti_reduction(const hipFloatComplex * beamformed,
		float * data_out) {

	int f = blockIdx.x;
	int b = blockIdx.y;
	int t = threadIdx.x;
	int s = blockIdx.z;

	float beam_power;
	float scale = 1.0/N_TIME_STI;

	//New variable//////
	__shared__ float reduced_array[N_STI_BLOC];
	/////////////////////

	if (t < N_TIME_STI) {
		hipFloatComplex samp = beamformed[sample_idx(s*N_TIME_STI+t,b,f)];
		//beam_power = (float)(hipCmulf(beamformed[sample_idx(s*N_TIME_STI+t,b,f)], hipConjf(beamformed[sample_idx(s*N_TIME_STI+t,b,f)])).x);
		beam_power = samp.x * samp.x + samp.y * samp.y;

		//atomicAdd(&data_out[output_idx(b,s,f)], beam_power*scale);
	}

	//New code///////////////////////////////////////////////

	if(t<N_TIME_STI){
		reduced_array[t] = beam_power;
	}
	else{
		reduced_array[t] = 0.0;
	}
	__syncthreads();

	for(int k = blockDim.x/2; k>0; k>>=1){
		if(t<k){
			reduced_array[t] += reduced_array[t+k];
		}
		__syncthreads();
	}
	if(t == 0){
		data_out[output_idx(b,s,f)] = reduced_array[0]*scale;
	}

	/////////////////////////////////////////////////////////
}


void run_beamformer(unsigned char * data, float * weights, float * out){
	// Specify grid and block dimensions
	dim3 dimBlock(N_STI_BLOC, 1, 1);
	dim3 dimGrid(N_BIN, N_BEAM, N_STI);

	dim3 dimBlock2(N_ELE_BLOC, 1, 1);
	dim3 dimGrid2(N_TIME, N_BIN, N_BEAM);
	
	unsigned char * d_data;
	hipFloatComplex * d_weights;
	hipFloatComplex * d_beamformed;
	float * d_outputs;

	hipMalloc((void **)&d_data, 2*N_SAMP*sizeof(unsigned char));
	hipMalloc((void **)&d_weights, N_WEIGHTS*sizeof(hipFloatComplex));
	hipMalloc((void **)&d_beamformed, N_TBF*sizeof(hipFloatComplex));
	hipMalloc((void **)&d_outputs, N_OUTPUTS*sizeof(float));

	hipMemcpyAsync(d_data, data, 2*N_SAMP*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(d_weights, weights, N_WEIGHTS*sizeof(hipFloatComplex), hipMemcpyHostToDevice);

	// Run the beamformer
	beamform<<<dimGrid2, dimBlock2>>>(d_data, d_weights, d_beamformed);
	hipError_t err_code = hipGetLastError();
	if (err_code != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(err_code));
	}

	sti_reduction<<<dimGrid, dimBlock>>>(d_beamformed, d_outputs);
	err_code = hipGetLastError();
	if (err_code != hipSuccess) {
		printf("CUDA Error (sti_reduction): %s\n", hipGetErrorString(err_code));
	}

	hipMemcpy(out, d_outputs, N_OUTPUTS*sizeof(float),
			hipMemcpyDeviceToHost);
	hipFree(d_data);
	hipFree(d_weights);
	hipFree(d_outputs);
	hipFree(d_beamformed);

}
