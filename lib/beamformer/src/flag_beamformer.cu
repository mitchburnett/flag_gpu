#include "hip/hip_runtime.h"
#include "flag_beamformer.h"
#include <hip/hip_complex.h>

/*
__global__
void beamform(const hipDoubleComplex * data_in,
              const hipDoubleComplex * weights,
              float * data_out) {

	int f = blockIdx.x;
	int b = blockIdx.y;
	int t = threadIdx.x;
	int s = blockIdx.z;
	//int e = threadIdx.y;

	int i = input_idx(s*N_TIME_STI + t,f,0);//int i = input_idx(t,f,e);
	int w = weight_idx(b,f,0);//int w = weight_idx(b,f,e);
	//int c = sample_idx(t,b,f);
	// ************ Complex multiplication *************
	hipDoubleComplex elem_val;
	hipDoubleComplex weight_val;
	hipDoubleComplex new_beam;
	float beam_power;

	int e; // Element index
	float scale = 1.0/N_TIME_STI;
	//New variables//////
	__shared__ float reduced_array[N_STI_BLOC];
	//reduced_array = (float *)malloc(N_TBF*sizeof(float));
	/////////////////////
	new_beam.x = 0.0;
	new_beam.y = 0.0;

	if (t < N_TIME_STI) {

		for (e = 0; e < N_ELE; e++) { // Loop over elements
			elem_val = data_in[i+e];
			weight_val = weights[w+e];
			new_beam = hipCadd(new_beam, hipCmul(elem_val, hipConj(weight_val)));
		}

		beam_power = (float)(hipCmul(new_beam, hipConj(new_beam)).x);
		//beam_power = (float) (hipCmul(data_in[i], hipConj(weights[w])).x);
		atomicAdd(&data_out[output_idx(b,s,f)], beam_power*scale);
	}

	//New code///////////////////////////////////////////////

	//if(t<N_TIME_STI){
	//	reduced_array[t] = beam_power;
	//}
	//else{
	//	reduced_array[t] = 0.0;
	//}
	//__syncthreads();
	//
	//for(int k = blockDim.x/2; k>0; k>>=1){
	//	if(t<k){
	//		reduced_array[t] += reduced_array[t+k];
	//	}
	//	__syncthreads();
	//}
	//if(t == 0){
	//	data_out[output_idx(b,s,f)] = reduced_array[0]*scale;
	//}

	/////////////////////////////////////////////////////////
}
 */



__global__
void beamform(const unsigned char * data_in,
		const hipFloatComplex * weights,
		hipFloatComplex * beamformed) {

	int e = threadIdx.x;
	int t = blockIdx.x;
	int f = blockIdx.y;
	int b = blockIdx.z;
	//

	int i = input_idx(t,f,e);
	int w = weight_idx(b,f,e);
	//int c = sample_idx(t,b,f);
	// ************ Complex multiplication *************
	//hipDoubleComplex elem_val;
	//hipDoubleComplex weight_val;
	//hipDoubleComplex new_beam;

	//float scale = 1.0/N_ELE;
	//New variables//////
	__shared__ hipFloatComplex reduced_mul[N_ELE_BLOC];
	/////////////////////
	//new_beam.x = 0.0;
	//new_beam.y = 0.0;

	//elem_val = data_in[i];
	//weight_val = weights[w];
	//new_beam = hipCmul(elem_val, hipConj(weight_val));

	//New code///////////////////////////////////////////////

	if(e<N_ELE) {
		reduced_mul[e].x = data_in[2*i]   * weights[w].x + data_in[2*i+1] * weights[w].y;
		reduced_mul[e].y = data_in[2*i+1] * weights[w].x - data_in[2*i]   * weights[w].y;
	}
	else {
		reduced_mul[e].x = 0;
		reduced_mul[e].y = 0;
	}
	__syncthreads();

	//atomicAdd(&(beamformed[sample_idx(t,b,f)].x),reduced_mul[e].x);
	//atomicAdd(&(beamformed[sample_idx(t,b,f)].y),reduced_mul[e].y);

	for(int k = blockDim.x/2; k>0; k>>=1){
		if(e<k){
			//reduced_mul[e] = hipCaddf(reduced_mul[e], reduced_mul[e+k]);
			reduced_mul[e].x = reduced_mul[e].x + reduced_mul[e+k].x;
			reduced_mul[e].y = reduced_mul[e].y + reduced_mul[e+k].y;
		}
		__syncthreads();
	}
	if(e == 0){
		beamformed[sample_idx(t,b,f)] = reduced_mul[0];
	}


	/////////////////////////////////////////////////////////
}


__global__
void sti_reduction(const hipFloatComplex * beamformed,
		float * data_out) {

	int f = blockIdx.x;
	int b = blockIdx.y;
	int t = threadIdx.x;
	int s = blockIdx.z;

	float beam_power;
	float scale = 1.0/N_TIME_STI;

	//New variable//////
	__shared__ float reduced_array[N_STI_BLOC];
	/////////////////////

	if (t < N_TIME_STI) {
		hipFloatComplex samp = beamformed[sample_idx(s*N_TIME_STI+t,b,f)];
		//beam_power = (float)(hipCmulf(beamformed[sample_idx(s*N_TIME_STI+t,b,f)], hipConjf(beamformed[sample_idx(s*N_TIME_STI+t,b,f)])).x);
		beam_power = samp.x * samp.x + samp.y * samp.y;

		//atomicAdd(&data_out[output_idx(b,s,f)], beam_power*scale);
	}

	//New code///////////////////////////////////////////////

	if(t<N_TIME_STI){
		reduced_array[t] = beam_power;
	}
	else{
		reduced_array[t] = 0.0;
	}
	__syncthreads();

	for(int k = blockDim.x/2; k>0; k>>=1){
		if(t<k){
			reduced_array[t] += reduced_array[t+k];
		}
		__syncthreads();
	}
	if(t == 0){
		data_out[output_idx(b,s,f)] = reduced_array[0]*scale;
	}

	/////////////////////////////////////////////////////////
}

static float * weights = NULL;
void init_beamformer() {
	weights = (float *)calloc(2*N_WEIGHTS, sizeof(float));
}

void update_weights(char * filename) {
	char weight_filename[128];
	strcpy(weight_filename, filename);
	FILE * weights_file;
	weights_file = fopen(weight_filename, "r");
	fread(weights, sizeof(float), 2*N_WEIGHTS, weights_file);
	fclose(weights_file);
}

void run_beamformer(unsigned char * data, float * out) {
	// Specify grid and block dimensions for both kernels
	dim3 dimBlock(N_STI_BLOC, 1, 1);
	dim3 dimGrid(N_BIN, N_BEAM, N_STI);

	dim3 dimBlock2(N_ELE_BLOC, 1, 1);
	dim3 dimGrid2(N_TIME, N_BIN, N_BEAM);
	
	// Device data pointers
	unsigned char * d_data;
	hipFloatComplex * d_weights;
	hipFloatComplex * d_beamformed;
	float * d_outputs;

	// Allocate memory on device
	hipMalloc((void **)&d_data, 2*N_SAMP2*sizeof(unsigned char));
	hipMalloc((void **)&d_weights, N_WEIGHTS*sizeof(hipFloatComplex));
	hipMalloc((void **)&d_beamformed, N_TBF*sizeof(hipFloatComplex));
	hipMalloc((void **)&d_outputs, N_OUTPUTS*sizeof(float));

	// Optional sanity check
	// int e = 0;
	// int f = 0;
	// int t = 0;
	// for (t = 0; t < 40; t++) {
	//	printf("data[t,f,e = %d,%d,%d] = %d + j%d\n", t, f, e, data[2*input_idx(t,f,e)], data[2*input_idx(t,f,e)+1]);
	// }

	// Copy data to device
	hipError_t err_code;
	err_code = hipMemcpy(d_data, data, 2*N_SAMP2*sizeof(unsigned char), hipMemcpyHostToDevice);
	if (err_code != hipSuccess) {
		printf("CUDA ERROR (cudaMemcpy1): %s\n", hipGetErrorString(err_code));
	}

	// Copy weights to device
	err_code = hipMemcpy(d_weights, weights, N_WEIGHTS*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
	if (err_code != hipSuccess) {
		printf("CUDA ERROR (cudaMemcpy2): %s\n", hipGetErrorString(err_code));
	}

	// Run the beamformer kernel
	beamform<<<dimGrid2, dimBlock2>>>(d_data, d_weights, d_beamformed);

	// Error checking
	err_code = hipGetLastError();
	if (err_code != hipSuccess) {
		printf("CUDA Error (beamform): %s\n", hipGetErrorString(err_code));
	}

	// Run the accumulation kernel
	sti_reduction<<<dimGrid, dimBlock>>>(d_beamformed, d_outputs);

	// Error checking
	err_code = hipGetLastError();
	if (err_code != hipSuccess) {
		printf("CUDA Error (sti_reduction): %s\n", hipGetErrorString(err_code));
	}

	// Copy data products to host from device
	hipMemcpy(out, d_outputs, N_OUTPUTS*sizeof(float),
			hipMemcpyDeviceToHost);

	// Free memory on device
	hipFree(d_data);
	hipFree(d_weights);
	hipFree(d_outputs);
	hipFree(d_beamformed);

}
