#include "beamformer_gpu.h"
#include <string.h>
#include <stdlib.h>
#include <time.h>

void printUsage();

int main(int argc, char * argv[]) {
	// Parse input
	if (argc != 4) {
		printUsage();
		return -1;
	}
	char input_filename[128];
	char weight_filename[128];
	char output_filename[128];

	strcpy(input_filename,  argv[1]);
	strcpy(weight_filename, argv[2]);
	strcpy(output_filename, argv[3]);

	// File pointers
	FILE * data;
	FILE * weights;
	
	// File data pointers
	float * bf_data;
	float * bf_weights;

	// Complex data pointers
	float complex * data_dc;
	float complex * weights_dc;

	// Allocate heap memory for file data
	bf_data = (float *)malloc(2*N_SAMP*sizeof(float));
	bf_weights = (float *)malloc(2*N_WEIGHTS*sizeof(float));
	data_dc = (float complex *)malloc(N_SAMP*sizeof(float complex *));
	weights_dc = (float complex *)malloc(N_WEIGHTS*sizeof(float complex *));

	// Open files
	data = fopen(input_filename, "r");
	weights = fopen(weight_filename, "r");

	// Read in data
	int j;
	if (data != NULL) {
		fread(bf_data, sizeof(float), 2*N_SAMP, data);

		// Make 'em complex!
		for (j = 0; j < N_SAMP; j++) {
			data_dc[j] = bf_data[2*j] + bf_data[(2*j)+1]*I;
		}
		fclose(data);
	}
	free(bf_data);

	if (weights != NULL) {
		fread(bf_weights, sizeof(float), 2*N_WEIGHTS, weights);
		// Make 'em complex!
		for (j = 0; j < N_WEIGHTS; j++) {
			weights_dc[j] = bf_weights[2*j] + bf_weights[(2*j)+1]*I;
		}
		fclose(weights);
	}
	free(bf_weights);

	// Allocate memory for the output
	float * output_f;
	output_f = (float *)calloc(N_OUTPUTS,sizeof(float));

	struct timespec tstart = {0,0};
	struct timespec tstop  = {0,0};
	clock_gettime(CLOCK_MONOTONIC, &tstart);

	// Specify grid and block dimensions
	dim3 dimBlock(N_STI_BLOC, 1, 1);
	dim3 dimGrid(N_BIN, N_BEAM, N_STI);
	
	dim3 dimBlock2(N_ELE_BLOC, 1, 1);
	dim3 dimGrid2(N_TIME, N_BIN, N_BEAM);

	hipFloatComplex * d_data;
	hipFloatComplex * d_weights;
	hipFloatComplex * d_beamformed;//////////
	float * d_outputs;

	//hipMalloc((void **)&d_data, N_SAMP*sizeof(hipDoubleComplex));
	//hipMalloc((void **)&d_weights, N_WEIGHTS*sizeof(hipDoubleComplex));
	//hipMalloc((void **)&d_outputs, N_OUTPUTS*sizeof(float));
	hipError_t err_malloc = hipMalloc((void **)&d_data, (N_SAMP + N_WEIGHTS)*sizeof(hipFloatComplex) + N_OUTPUTS*sizeof(float));
	if (err_malloc != hipSuccess) {
		printf("CUDA Error (cudaMalloc1): %s\n", hipGetErrorString(err_malloc));
	}
	err_malloc = hipMalloc((void **)&d_beamformed, N_TBF*sizeof(hipFloatComplex));
	if (err_malloc != hipSuccess) {
		printf("CUDA Error (cudaMalloc2): %s\n", hipGetErrorString(err_malloc));
	}

	d_weights = d_data + N_SAMP;
	d_outputs = (float *)(d_data + N_SAMP + N_WEIGHTS);
	hipMemset(d_outputs, 0.0, N_OUTPUTS*sizeof(float));
	
	//printf("data_dc weights_dc %.7e %e\n",data_dc,weights_dc);
	hipMemcpy(d_data,    data_dc,   N_SAMP*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_weights, weights_dc, N_WEIGHTS*sizeof(hipFloatComplex), hipMemcpyHostToDevice);


	//printf("data_dc:\t%.7e+%.7e*I\n weights_dc:\t%.7e+%.7e*I\n",data_dc[0],weights_dc[0]);

	// Run the beamformer
	//printf("D_data D_weights %.7e + %.7e*I\n",temp);
	printf("Starting beamformer\n");
	beamform<<<dimGrid2, dimBlock2>>>(d_data, d_weights, d_beamformed);//beamform<<<dimGrid, dimBlock>>>(d_data, d_weights, d_beamformed);
	//printf("D_data D_weights D_outputs %.7e %e %e\n",d_data,d_weights,d_outputs);
	printf("Finishing beamformer\n");

	hipError_t err_code = hipGetLastError();
	if (err_code != hipSuccess) {
		printf("CUDA Error (beamform): %s\n", hipGetErrorString(err_code));
	}

	//printf("Beamformed %e+%e*I\n", temp);

	printf("Starting sti_reduction\n");
	sti_reduction<<<dimGrid, dimBlock>>>(d_beamformed,d_outputs);
	printf("Finishing sti_reduction\n");

	err_code = hipGetLastError();
	if (err_code != hipSuccess) {
		printf("CUDA Error (sti_reduction): %s\n", hipGetErrorString(err_code));
	}
	
	
	hipMemcpy(output_f, d_outputs, N_OUTPUTS*sizeof(float), hipMemcpyDeviceToHost);
	//printf("Output %e\n",output_f[0]);
	hipFree(d_data);
	hipFree(d_weights);
	hipFree(d_outputs);

	clock_gettime(CLOCK_MONOTONIC, &tstop);
	//printf("Beamformer elapsed time: %.5f seconds\n",
		//((double)tstop.tv_sec + 1.0e-9*tstop.tv_nsec) -
		//((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec));
	
	// Save output data to file
	FILE * output;
	output = fopen(output_filename, "w");
	fwrite(output_f, sizeof(float), N_OUTPUTS, output);
	fclose(output);

	free(data_dc);
	free(weights_dc);
	free(output_f);

	return 0;
}

void printUsage() {
	printf("Usage: my_beamformer <input_filename> <weight_filename> <output_filename>\n");
}

//For makefile at the very end "-fno-exceptions -fno-rtti"
